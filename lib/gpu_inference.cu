#include "hip/hip_runtime.h"
#include "gpu_inference.cuh"

#include "hip/hip_runtime.h"
#include ""

#include "ImageData.h"
#include "weight_loader.h"
#include <stdio.h>
#include <array>
#include <cmath>
#include <cassert>

#define CDIV(X, Y) (((X) + (Y) - 1) / (Y))

#define BLK_SZ 16

hipEvent_t finishEvt = NULL;

__global__ void linearKernel(float* X, float* W, float* B, float* out,
                              int M, int K, int N, bool relu) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ float sX[BLK_SZ][BLK_SZ];
    __shared__ float sW[BLK_SZ][BLK_SZ];

    int localCol = threadIdx.x;
    int localRow = threadIdx.y;

    float acc = 0.f;

    int bSize = CDIV(K, BLK_SZ);

    for (int b = 0; b < bSize; ++b) {
        int offs = b * BLK_SZ;

        if (row >= M || offs + localCol >= K) {
            sX[localRow][localCol] = 0.f;
        } else {
            sX[localRow][localCol] = X[row * K + (offs + localCol)];
        }

        if (col >= N || offs + localRow >= K) {
            sW[localRow][localCol] = 0.f;
        } else {
            sW[localRow][localCol] = W[(offs + localRow) * N + col];
        }

        __syncthreads();

        for (int k = 0; k < BLK_SZ; ++k) {
            acc += sX[localRow][k] * sW[k][localCol];
        }

        __syncthreads();
    }

    if (row >= M || col >= N)
        return;

    float ret = acc + B[col];

    if (relu && ret < 0.f) {
        ret = 0.f;
    }

    out[row * N + col] = ret;
}


__global__ void matchCountKernel(float* X, int8_t* labels, int* count, int M, int N, int imgIdx) {
    int row = threadIdx.x;
    int retIdx = row * N;
    int retLabel = 0;

    for (int i = 1; i < N; ++i) { //10 shared
        int idx = row * N + i;
        if (X[retIdx] < X[idx]) {
            retIdx = idx;
            retLabel = i;
        }
    }

    if (retLabel == labels[imgIdx + row]) {
        atomicAdd(count, 1); //reduction
    }
}


template <typename T>
void memcpyGPU(T** p, const T* data, int memSz) {
    hipMalloc(p, memSz);
    hipMemset(*p, 0, memSz);
    hipMemcpy(*p, data, memSz, hipMemcpyHostToDevice);
    ASSERT_CUDA;
}

template <typename T>
void memsetGPU(T** p, int memSz) {
    hipMalloc(p, memSz);
    hipMemset(*p, 0, memSz);
    ASSERT_CUDA;
};

InferenceManager::InferenceManager(Model* model, int inpSz, int numBt,
                                   const std::vector<int8_t>& labels) : m_model(model), m_inpSz(inpSz), m_numBt(numBt) {
    auto& linears = model->linears;

    m_mkn.push_back({numBt, inpSz, linears[0].col});

    for (int i = 1; i < linears.size(); ++i) {
        m_mkn.push_back({numBt, linears[i-1].col, linears[i].col});
    }

    for (int i = 0; i < linears.size(); ++i) {
        int M = m_mkn[i].m;
        int K = m_mkn[i].k;
        int N = m_mkn[i].n;

        float *W, *B, *out;
        memcpyGPU(&W, linears[i].w.data(), sizeof(float) * N * K);
        memcpyGPU(&B, linears[i].b.data(), sizeof(float) * K);
        memsetGPU(&out, sizeof(float) * M * N);

        m_wBuffers.push_back(W);
        m_bBuffers.push_back(B);
        m_outBuffers.push_back(out);
    }

    memsetGPU(&m_inpBuffer, sizeof(float) * m_numBt * m_inpSz);
    memcpyGPU(&m_labelBuffer, labels.data(), sizeof(int8_t) * labels.size());
    memsetGPU(&m_pCnt, sizeof(int));
}


void InferenceManager::inferenceOnGPU(ImageData& img, int imgIdx, std::vector<int8_t>& labels) {

    auto& linears = m_model->linears;

    hipMemcpy(m_inpBuffer, img.data.data() + (imgIdx * m_inpSz),
               sizeof(float) * m_numBt * m_inpSz, hipMemcpyHostToDevice);
    ASSERT_CUDA;

    for (int i = 0; i < linears.size(); ++i) {
        int M = m_mkn[i].m;
        int K = m_mkn[i].k;
        int N = m_mkn[i].n;

        float* X = (i == 0) ? m_inpBuffer : m_outBuffers[i-1];
        float* W = m_wBuffers[i];
        float* B = m_bBuffers[i];
        float* out = m_outBuffers[i];

        dim3 gridDim(CDIV(N, BLK_SZ), CDIV(M, BLK_SZ)); //y-row, x-col
        dim3 blockDim(BLK_SZ, BLK_SZ);
        linearKernel <<<gridDim,  blockDim>>>(X, W, B, out, M, K, N, i < linears.size() - 1);
        ASSERT_CUDA;
    }

    dim3 gridDim(1);
    dim3 blockDim(m_numBt);
    matchCountKernel <<<gridDim, blockDim>>>(m_outBuffers.back(), m_labelBuffer, m_pCnt, m_numBt, 10, imgIdx);

    bool isLastOperation = (imgIdx == 10000 - m_numBt);
    if (isLastOperation) {
        hipEventCreate(&finishEvt);
        hipEventRecord(finishEvt);
    }
}

bool InferenceManager::checkFinish() {
    assert(finishEvt != NULL);
    bool isFinish = hipEventQuery(finishEvt) == hipSuccess;

    if (isFinish) {
        int cntHost = 0;
        hipMemcpy(&cntHost, m_pCnt, sizeof(int), hipMemcpyDeviceToHost);
        m_matchCount = cntHost;
    }

    return isFinish;
}

InferenceManager::~InferenceManager() {
    for (int i = 0; i < m_wBuffers.size(); ++i) {
        hipFree(m_wBuffers[i]);
        hipFree(m_bBuffers[i]);
        hipFree(m_outBuffers[i]);
    }

    if (m_inpBuffer) {
        hipFree(m_inpBuffer);
    }

    if (m_labelBuffer) {
        hipFree(m_labelBuffer);
    }

    if (m_labelBuffer) {
        hipFree(m_labelBuffer);
    }

    if (m_pCnt) {
        hipFree(m_pCnt);
    }

    hipEventDestroy(finishEvt);
    finishEvt = NULL;
}